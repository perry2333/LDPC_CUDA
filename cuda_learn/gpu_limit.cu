#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    printf("Device Name: %s\n", prop.name);
    printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("Max Threads Per Block: %d\n", prop.maxThreadsPerBlock);
    printf("Max Threads Per Multiprocessor: %d\n", prop.maxThreadsPerMultiProcessor);
    printf("Max Blocks Per Grid: (%d, %d, %d)\n", 
           prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("Multiprocessor Count: %d\n", prop.multiProcessorCount);
    printf("Warp Size: %d\n", prop.warpSize);
    
    return 0;
}
